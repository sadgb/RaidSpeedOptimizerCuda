#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <string>
#include <inttypes.h>

enum ChampionType {
    ApothecaryType,
    WarcasterType,
    KymerType,
    SkullcrusherType,
    RenegateType,
    OtherType
};


//#define GPU_PRINT(x) printf(x)

#define GPU_PRINT(x) 




struct Entity {
    int turnMeter = 0;
    int speed;
    int speed30Duration = 0;
};

struct ChampionStruct : Entity {  // Structure declaration  
    int_fast8_t skillCooldown = 0;
    int_fast8_t skillCurrentCooldown = 0;
    int_fast8_t skillDelay = 0;
    int_fast8_t unkillableDuration = 0;
    ChampionType type;
};

ChampionStruct* Apothecary() {
    ChampionStruct* result = new ChampionStruct();
    result->type = ChampionType::ApothecaryType;
    result->skillCooldown = 3;
    return result;
}


ChampionStruct* Warcaster() {
    ChampionStruct* result = new ChampionStruct();
    result->type = ChampionType::WarcasterType;
    result->skillCooldown = 4;
    return result;
}

ChampionStruct* Kymer() {
    ChampionStruct* result = new ChampionStruct();
    result->type = ChampionType::KymerType;
    result->skillCooldown = 6;
    return result;
}


ChampionStruct* Other() {
    ChampionStruct* result = new ChampionStruct();
    result->type = ChampionType::OtherType;
    return result;
}

struct ClanBoss : Entity {
    int turnesMade = 0;
};

struct Simulation {
    ChampionStruct c1, c2, c3, c4, c5;
    ClanBoss cb;
};

struct SimulationParams {
    int c1StartSpeed;
    int c1SpeedSteps;
    int c1SkillDelayMin;
    int c1SkillDelaySteps;

    int c2StartSpeed;
    int c2SpeedSteps;
    int c2SkillDelayMin;
    int c2SkillDelaySteps;

    int c3StartSpeed;
    int c3SpeedSteps;
    int c3SkillDelayMin;
    int c3SkillDelaySteps;

    int c4StartSpeed;
    int c4SpeedSteps;
    int c4SkillDelayMin;
    int c4SkillDelaySteps;

    int c5StartSpeed;
    int c5SpeedSteps;
    int c5SkillDelayMin;
    int c5SkillDelaySteps;

    int cbSpeed = 190;
};


SimulationParams* GetSimulationParams() {
    SimulationParams* params = new SimulationParams;
    params->c1StartSpeed = 200;
    params->c1SpeedSteps = 96;
    params->c1SkillDelayMin = 0;
    params->c1SkillDelaySteps = 2;
    //----------
    params->c2StartSpeed = 200;
    params->c2SpeedSteps = 80;
    params->c2SkillDelayMin = 0;
    params->c2SkillDelaySteps = 4;
    //----------
    params->c3StartSpeed = 200;
    params->c3SpeedSteps = 96;
    params->c3SkillDelayMin = 0;
    params->c3SkillDelaySteps = 5;
    //----------
    params->c4StartSpeed = 200;
    params->c4SpeedSteps = 0;
    params->c4SkillDelayMin = 0;
    params->c4SkillDelaySteps = 0;
    //----------
    params->c5StartSpeed = 200;
    params->c5SpeedSteps = 0;
    params->c5SkillDelayMin = 0;
    params->c5SkillDelaySteps = 0;

    return params;
}
/**/
/**/

/*
// working example
// Fastest speed tuned team had speeds : 302, 286, 286, 270, 200
// delays : d1 = 1 d2 = 0 d3 = 2
SimulationParams* GetSimulationParams() {
    SimulationParams* params = new SimulationParams;
    params->c1StartSpeed = 302;
    params->c1SpeedSteps = 0;
    params->c1SkillDelayMin = 1;
    params->c1SkillDelaySteps = 0;
    //----------
    params->c2StartSpeed = 286;
    params->c2SpeedSteps = 0;
    params->c2SkillDelayMin = 0;
    params->c2SkillDelaySteps = 0;
    //----------
    params->c3StartSpeed = 286;
    params->c3SpeedSteps = 0;
    params->c3SkillDelayMin = 2;
    params->c3SkillDelaySteps = 0;
    //----------
    params->c4StartSpeed = 270;
    params->c4SpeedSteps = 0;
    params->c4SkillDelayMin = 0;
    params->c4SkillDelaySteps = 0;
    //----------
    params->c5StartSpeed = 200;
    params->c5SpeedSteps = 0;
    params->c5SkillDelayMin = 0;
    params->c5SkillDelaySteps = 0;

    return params;
}
/**/
// N is the maximum number of structs to insert
#define N 10000
#define MAX_TURN_METER 1428.57


__device__ uint64_t dev_data[N];
__device__ int dev_count = 0;
__device__ int dev_founded = 0;

__device__ int my_push_back(uint64_t mt) {
    if (dev_count < N-10) {
        int insert_pt = atomicAdd(&dev_count, 1);
        if (insert_pt < N) {
            dev_data[insert_pt] = mt;
            return insert_pt;
        }
        else return -1;
    }
}

uint64_t CalculateSimulationParamsVariations(SimulationParams* params) {
    uint64_t result = (params->c1SpeedSteps + 1) * (params->c1SkillDelaySteps + 1);
    result *= (params->c2SpeedSteps + 1) * (params->c2SkillDelaySteps + 1) *
        (params->c3SpeedSteps + 1) * (params->c3SkillDelaySteps + 1) *
        (params->c4SpeedSteps + 1) * (params->c4SkillDelaySteps + 1) *
        (params->c5SpeedSteps + 1) * (params->c5SkillDelaySteps + 1);

    return result;
}

hipError_t testWithCuda(Simulation* simulation, SimulationParams* params);

__device__ void tickTurnmeter(Entity* e) {
    // TODO other speed bufs / debufs
    if (e->speed30Duration > 0) {
        e->turnMeter += e->speed * 1.3;
    }
    else {
        e->turnMeter += e->speed;
    }
}

__device__ void tickAllTurnmeters(Simulation* s) {
    tickTurnmeter(&s->c1);
    tickTurnmeter(&s->c2);
    tickTurnmeter(&s->c3);
    tickTurnmeter(&s->c4);
    tickTurnmeter(&s->c5);
    tickTurnmeter(&s->cb);
}

__device__ bool makeClanBossTurn(Simulation* s) {
    s->cb.turnesMade++;
    s->cb.turnMeter = 0;
    GPU_PRINT("\tBOSS\n");

    if (s->cb.turnesMade > 3 && (s->cb.turnesMade % 3 == 1 || s->cb.turnesMade % 3 == 2)) {
        // starting checks on 4th turn. only when it is AOE 1-2
        return s->c1.unkillableDuration > 0 &&
            s->c2.unkillableDuration > 0 &&
            s->c3.unkillableDuration > 0 &&
            s->c4.unkillableDuration > 0 &&
            s->c5.unkillableDuration > 0;
    }

    return true;
}

__device__ void makeChampionTurn(Simulation* s, ChampionStruct* c) {
    c->turnMeter = 0;
    c->unkillableDuration--;
    c->speed30Duration--;
    // TODO other buffs

    // TMP log move
    switch (c->type)
    {
    case ApothecaryType:
        GPU_PRINT(" A");
        break;
    case WarcasterType:
        GPU_PRINT(" W");
        break;
    case KymerType:
        GPU_PRINT(" K");
        break;
    default:
        GPU_PRINT("-");
    }

    if (c->skillDelay <= 0 && c->skillCurrentCooldown <= 0) {
        // perform skill
        switch (c->type)
        {
        case ApothecaryType:
            GPU_PRINT("S");
            // Fills the Turn Meter of all allies by 15 %.
            s->c1.turnMeter += MAX_TURN_METER * 15 / 100;
            s->c2.turnMeter += MAX_TURN_METER * 15 / 100;
            s->c3.turnMeter += MAX_TURN_METER * 15 / 100;
            s->c4.turnMeter += MAX_TURN_METER * 15 / 100;
            s->c5.turnMeter += MAX_TURN_METER * 15 / 100;
            // Places a 30 % Increase Speed buff on all allies for 2 turns
            s->c1.speed30Duration = 2;
            s->c2.speed30Duration = 2;
            s->c3.speed30Duration = 2;
            s->c4.speed30Duration = 2;
            s->c5.speed30Duration = 2;
            break;
        case WarcasterType:
            GPU_PRINT("S");
            //  Places block damage on all allies
            s->c1.unkillableDuration = 1;
            s->c2.unkillableDuration = 1;
            s->c3.unkillableDuration = 1;
            s->c4.unkillableDuration = 1;
            s->c5.unkillableDuration = 1;
            break;
        case KymerType:
            GPU_PRINT("S");
            // Fills the Turn Meter of all allies by 20 %.

            if (&s->c1 != c) { s->c1.turnMeter += MAX_TURN_METER * 20 / 100; }
            if(&s->c2 != c) { s->c2.turnMeter += MAX_TURN_METER * 20 / 100; }
            if(&s->c3 != c) { s->c3.turnMeter += MAX_TURN_METER * 20 / 100; }
            if(&s->c4 != c) { s->c4.turnMeter += MAX_TURN_METER * 20 / 100; }
            if(&s->c5 != c) { s->c5.turnMeter += MAX_TURN_METER * 20 / 100; }


            // Resets the cooldown of ally skills
            // since we will put our skill on cooldown later, it's ok to reset our skill as well
            s->c1.skillCurrentCooldown = 0;
            s->c2.skillCurrentCooldown = 0;
            s->c3.skillCurrentCooldown = 0;
            s->c4.skillCurrentCooldown = 0;
            s->c5.skillCurrentCooldown = 0;


            break;
        default:
            break;
        }

        // put the skill on cooldown
        c->skillCurrentCooldown = c->skillCooldown;
    }

    c->skillCurrentCooldown--;
    c->skillDelay--;
}


__device__ bool makeTurn(Simulation* s) {

    if (s->cb.turnMeter >= MAX_TURN_METER ||
        s->c1.turnMeter >= MAX_TURN_METER ||
        s->c2.turnMeter >= MAX_TURN_METER ||
        s->c3.turnMeter >= MAX_TURN_METER ||
        s->c4.turnMeter >= MAX_TURN_METER ||
        s->c5.turnMeter >= MAX_TURN_METER)
    {
        if (s->c1.turnMeter >= s->cb.turnMeter &&
            s->c1.turnMeter >= s->c2.turnMeter &&
            s->c1.turnMeter >= s->c3.turnMeter &&
            s->c1.turnMeter >= s->c4.turnMeter &&
            s->c1.turnMeter >= s->c5.turnMeter)
        {
            makeChampionTurn(s, &s->c1);
        }
        else if (s->c2.turnMeter >= s->cb.turnMeter &&
            s->c2.turnMeter >= s->c1.turnMeter &&
            s->c2.turnMeter >= s->c3.turnMeter &&
            s->c2.turnMeter >= s->c4.turnMeter &&
            s->c2.turnMeter >= s->c5.turnMeter)
        {
            makeChampionTurn(s, &s->c2);
        }
        else if (s->c3.turnMeter >= s->cb.turnMeter &&
            s->c3.turnMeter >= s->c1.turnMeter &&
            s->c3.turnMeter >= s->c2.turnMeter &&
            s->c3.turnMeter >= s->c4.turnMeter &&
            s->c3.turnMeter >= s->c5.turnMeter)
        {
            makeChampionTurn(s, &s->c3);
        }
        else if (s->c4.turnMeter >= s->cb.turnMeter &&
            s->c4.turnMeter >= s->c1.turnMeter &&
            s->c4.turnMeter >= s->c2.turnMeter &&
            s->c4.turnMeter >= s->c3.turnMeter &&
            s->c4.turnMeter >= s->c5.turnMeter)
        {
            makeChampionTurn(s, &s->c4);
        }
        else if (s->c5.turnMeter >= s->cb.turnMeter &&
            s->c5.turnMeter >= s->c1.turnMeter &&
            s->c5.turnMeter >= s->c2.turnMeter &&
            s->c5.turnMeter >= s->c3.turnMeter &&
            s->c5.turnMeter >= s->c4.turnMeter)
        {
            makeChampionTurn(s, &s->c5);
        }
        else if (s->cb.turnMeter >= s->c1.turnMeter &&
            s->cb.turnMeter >= s->c2.turnMeter &&
            s->cb.turnMeter >= s->c3.turnMeter &&
            s->cb.turnMeter >= s->c4.turnMeter &&
            s->cb.turnMeter >= s->c5.turnMeter)
        {
            return makeClanBossTurn(s);
        }
    }
    return true;

}

__device__  uint64_t getGlobalIdx() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.z * (blockDim.x * blockDim.y))
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}


__device__ __host__ void FillSpeedsFromIndex(Simulation* s, SimulationParams* params, uint64_t i) {

    // initialize speeds

    s->c1.speed = params->c1StartSpeed + i % (params->c1SpeedSteps + 1);
    i = i / (params->c1SpeedSteps + 1);
    s->c1.skillDelay = params->c1SkillDelayMin + i % (params->c1SkillDelaySteps + 1);
    i = i / (params->c1SkillDelaySteps + 1);

    s->c2.speed = params->c2StartSpeed + i % (params->c2SpeedSteps + 1);
    i = i / (params->c2SpeedSteps + 1);
    s->c2.skillDelay = params->c2SkillDelayMin + i % (params->c2SkillDelaySteps + 1);
    i = i / (params->c2SkillDelaySteps + 1);


    s->c3.speed = params->c3StartSpeed + i % (params->c3SpeedSteps + 1);
    i = i / (params->c3SpeedSteps + 1);
    s->c3.skillDelay = params->c3SkillDelayMin + i % (params->c3SkillDelaySteps + 1);
    i = i / (params->c3SkillDelaySteps + 1);

    s->c4.speed = params->c4StartSpeed + i % (params->c4SpeedSteps + 1);
    i = i / (params->c4SpeedSteps + 1);
    s->c4.skillDelay = params->c4SkillDelayMin + i % (params->c4SkillDelaySteps + 1);
    i = i / (params->c4SkillDelaySteps + 1);


    s->c5.speed = params->c5StartSpeed + i % (params->c5SpeedSteps + 1);
    i = i / (params->c5SpeedSteps + 1);
    s->c5.skillDelay = params->c5SkillDelayMin + i % (params->c5SkillDelaySteps + 1);

    s->cb.speed = params->cbSpeed;
}

__global__ void test(Simulation* simulation, SimulationParams* params) {

    Simulation s;

    memcpy(&s, simulation, sizeof(Simulation));

    FillSpeedsFromIndex(&s, params, getGlobalIdx());

    /*
    printf("%d:%d\n%d:%d\n%d:%d\n%d:%d\n%d:%d\n", s.c1.speed, s.c1.skillDelay,
        s.c2.speed, s.c2.skillDelay, 
        s.c3.speed, s.c3.skillDelay, 
        s.c4.speed, s.c4.skillDelay, 
        s.c5.speed, s.c5.skillDelay);

    printf("===%d====\n\n", i);
    /**/
    bool running = true;
    
    while (running) {
        tickAllTurnmeters(&s);
        running = makeTurn(&s);
        if (s.cb.turnesMade >= 50) {
            // STOP after 50 turns
            running = false;
        }
    }
    /**/
    
    if (s.cb.turnesMade >= 50) {
        // SUCCESS
        my_push_back(getGlobalIdx());
       atomicAdd(&dev_founded, 1);
    }
    /**/
}


int main()
{
    const int arraySize = 5;
    int result[arraySize];

    Simulation* x = new Simulation;
    memcpy(&(x->c1), Warcaster(), sizeof(ChampionStruct));
    memcpy(&(x->c2), Apothecary(), sizeof(ChampionStruct));
    memcpy(&(x->c3), Kymer(), sizeof(ChampionStruct));
    memcpy(&(x->c4), Other(), sizeof(ChampionStruct));
    memcpy(&(x->c5), Other(), sizeof(ChampionStruct));

    SimulationParams* params = GetSimulationParams();



    

    // Add vectors in parallel.
    hipError_t cudaStatus = testWithCuda(x, params);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "testWithCuda failed!");
        return 1;
    }


    //printf("{%d,%d,%d,%d,%d}\n",
    //    result[0], result[1], result[2], result[3], result[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t testWithCuda(Simulation* simulation, SimulationParams* params)
{
    SimulationParams* gpuSimulationParams;
    Simulation* gpuSimulation;
   // int *dev_b = 0;
    int *gpu_result = 0;
    hipError_t cudaStatus;


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&gpuSimulationParams, sizeof(SimulationParams));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&gpuSimulation, sizeof(Simulation));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // TODO size
    cudaStatus = hipMalloc((void**)&gpu_result, 5 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(gpuSimulationParams, params, sizeof(SimulationParams), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    cudaStatus = hipMemcpy(gpuSimulation, simulation, sizeof(Simulation), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    /*
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }


    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    */

    uint64_t variations = CalculateSimulationParamsVariations(params);
    int block_size = 128;
    uint64_t blocks_count =  variations / block_size;
    //int blocks_count = 1;
    fprintf(stderr, "CalculateSimulationParamsVariations %" PRIu64 "kk,\nblock_size=%i\nblocks_count=%ukk\nestimated_time=%.1fs\n", variations/1000000, block_size, blocks_count / 1000000, variations*1.0 / 300000000);


   

    const clock_t begin_time = clock();

    // Launch a kernel on the GPU with one thread for each element.
    test <<<blocks_count, block_size >>>(gpuSimulation, gpuSimulationParams);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching CUDA!\n", cudaStatus);
        goto Error;
    }


    const clock_t end_time = clock();

    float seconds = (end_time - begin_time) * 1.0 / CLOCKS_PER_SEC;
    float ips = variations * 1.0 / seconds / 1000000;

    printf("Finished. Total time: %.1fs, speed: %.1fkk per second\n", seconds, ips);


    int founded;
    cudaStatus = hipMemcpyFromSymbol(&founded, HIP_SYMBOL(dev_founded), sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA hipMemcpyFromSymbol failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    printf("founded=%d\n", founded);

    int dsize;
    cudaStatus = hipMemcpyFromSymbol(&dsize, HIP_SYMBOL(dev_count), sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA hipMemcpyFromSymbol failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    printf("gpuResult.count=%d\n", dsize);


    uint64_t result[N];
    cudaStatus = hipMemcpyFromSymbol(&result, HIP_SYMBOL(dev_data), N * sizeof(uint64_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA hipMemcpyFromSymbol result failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    Simulation s;
    for (int i = 0; i < 10; i++) {
        FillSpeedsFromIndex(&s, params, result[i]);
        printf("FOUND:  C1=%d:%d C2=%d:%d C3=%d:%d C4=%d:%d C5=%d:%d\n", s.c1.speed, s.c1.skillDelay
            , s.c2.speed, s.c2.skillDelay, s.c3.speed, s.c3.skillDelay, s.c4.speed, s.c4.skillDelay, s.c5.speed, s.c5.skillDelay);
    }

    


    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(result, gpu_result, 5, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }


Error:
    hipFree(gpuSimulation);
    hipFree(gpuSimulationParams);
    
    return cudaStatus;
}
